
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime_api.h>

#define SQ(x) ((x) * (x))

static const float A = -4.0, B = 4.0;  // limites de integración
static const int N = 1 << 22;          // número de intervalos = 2^22
static const float H = (B - A) / N;    // tamaño del intervalo de integración
static const float PI(M_PI);           // π con precision simple

float h(float x) {
    return .5f + 1.5f / (1.0f + 50.0f * SQ(x));
}

float f(float x) {
    int i;
    float sum = 0.0f, x0;
    for (i = 0; i < 10; ++i)
        x0 = -3.3f + i * 0.7f;
        sum += h(x - x0);
    return sum/10.0f;
}

float g(float x) {
    float c = cosf(2.0f * PI * f(x) * x);
    return expf(-x/16.0f) * SQ(c);
}

__global__ void
integrate_blocks(float subtotals[]) {


}


int main(int argc, char *argv[]) {

    // El programa recibe como parámetro el número de hebras por bloque.
    // Recuerden que este número debe ser múltiplo de 32 (tamaño del warp)
    // y puede ser a lo más 512 (limitación del hardware).

    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " threads_per_block" << std::endl;
        std::exit(1);
    }
    int block_size = std::atoi(argv[1]);

    // Al usar N subintervalos, hay que evaluar la función en 2N + 1 puntos.
    // Para paralelizar, mejor evaluar 2N puntos y sumar el último al final.
    // Por lo tanto, necesitamos 2N hebras.
    int nr_blocks = 2 * N / block_size;

    // Reservar arreglos en RAM y en la GPU para guardar los resultados.
    float *subtotals_h, *subtotals_d;
    subtotals_h = new float[nr_blocks];
    hipMalloc((void **) &subtotals_d, sizeof(float) * nr_blocks);

    integrate_blocks<<<nr_blocks, block_size>>>(subtotals_d);

    // En la parte (a) de la tarea,
    // la reducción global la hacemos en la CPU.

    hipMemcpy(subtotals_h, subtotals_d,
               sizeof(float) * nr_blocks, hipMemcpyDeviceToHost);

    float sum = 0.0;
#   pragma omp parallel for reduction(+: sum)
    for (int b = 0; b < nr_blocks; ++b) {
        sum += subtotals_h[b];
    }

    std::cout << sum << std::endl;

    hipFree(subtotals_d);
    std::free(subtotals_h);
}

