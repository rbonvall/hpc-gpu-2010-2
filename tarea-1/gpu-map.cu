#include "hip/hip_runtime.h"
#include "gpu-map.hpp"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>

// Los kernels d_f y d_g implementan las funciones f y g
// aplicadas a un valor x cualquiera.
// Estos kernels están declarados con el calificador __device__,
// por lo que sólo pueden ser llamados por código
// que está siendo ejecutado en la GPU.

__device__ float d_f(float x) {
    float s = 0.0;
    for (int k = 1; k <= 10000; ++k) {
        s += sinf(2 * float(M_PI) * k * x);
    }
    return s;
}
__device__ float d_g(float x) {
    return x * x;
}

// Los kernels map_f y map_g se encargan de que cada hebra
// ejecute las funciones d_f y d_g con el argumento apropiado.

__global__ void map_f(float x[]) {
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    x[i] = d_f(x[i]);
}

__global__ void map_g(float x[]) {
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    x[i] = d_g(x[i]);
}


// gpu_map hace toda la burocracia de ejecutar código en la GPU:
// reserva memoria, copia el arreglo, lanza el kernel,
// copia el resultado de vuelta y libera la memoria.
//
// La función está parametrizada con el nombre de la función
// representado como un char. Es posible encapsular los kernels
// usando functores para usar plantillas, pero ahora preferí
// que el código CUDA fuera lo más simple posible.

void gpu_map(char function_name, float x[], unsigned n) {

    unsigned mem_size = sizeof(float) * n;
     
    float *d_x;
    hipMalloc((void **) &d_x, n * sizeof(float));
    hipMemcpy(d_x, x, mem_size, hipMemcpyHostToDevice);

    dim3 grid_size, block_size;
    block_size.x = 512;      // number of threads per block (<= 512)
    grid_size.x = n / 512;   // number of blocks

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    switch (function_name) {
        case 'f': map_f<<<grid_size, block_size>>>(d_x); break;
        case 'g': map_g<<<grid_size, block_size>>>(d_x); break;
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemcpy(x, d_x, mem_size, hipMemcpyDeviceToHost);
    hipFree(d_x);
    d_x = 0;
}

