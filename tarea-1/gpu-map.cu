#include "hip/hip_runtime.h"
#include "gpu-map.hpp"
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdio.h>

// El kernel d_f implementa la función f aplicadas a un valor x cualquiera.
// Este kernel está declarado con el calificador __device__,
// por lo que sólo puede ser llamado por código que está siendo ejecutado
// en la GPU.

__device__ float d_f(float x) {
    float s = 0.0;
    for (int k = 1; k <= 10000; ++k) {
        s += sinf(2 * float(M_PI) * k * x);
    }
    return s;
}

// El kernel map_f se encarga de que cada hebra
// ejecute la función d_f con el argumento apropiado.

__global__ void map_f(float x[]) {
    unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
    x[i] = d_f(x[i]);
}

// gpu_map debe hacer toda la burocracia de ejecutar código en la GPU:
// reserva memoria, copia el arreglo, lanza el kernel,
// copia el resultado de vuelta y libera la memoria.

void gpu_map(float x[], unsigned n) {

    unsigned mem_size = sizeof(float) * n;
     
    float *d_x;
    hipMalloc((void **) &d_x, n * sizeof(float));
    hipMemcpy(d_x, x, mem_size, hipMemcpyHostToDevice);

    dim3 grid_size, block_size;
    block_size.x = 512;      // number of threads per block (<= 512)
    grid_size.x = n / 512;   // number of blocks

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    map_f<<<grid_size, block_size>>>(d_x);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    hipMemcpy(x, d_x, mem_size, hipMemcpyDeviceToHost);
    hipFree(d_x);
    d_x = 0;
}

